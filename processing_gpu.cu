#include "hip/hip_runtime.h"
/*
    USRP_Software_defined_radar is a software for real time sampling, processing, display and storing
    Copyright (C) 2018  Jonas Myhre Christiansen <jonas-myhre.christiansen@ffi.no>
	
    This file is part of USRP_Software_defined_radar.

    USRP_Software_defined_radar is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    USRP_Software_defined_radar is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with USRP_Software_defined_radar.  If not, see <https://www.gnu.org/licenses/>.
*/

#include "processing_gpu.h"

__global__ void
rdComplexMultiply(hipFloatComplex *s, hipFloatComplex *w, long int M, long int N)
{
    long int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N*M)
    {
        long int n = i % N;

        s[i] = hipCmulf(s[i], hipConjf(w[n]));
    }
}

__global__ void
rdComplexTranspose(hipFloatComplex *sout, hipFloatComplex *sin, long int M, long int N)
{
    long int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N*M)
    {
        long int n = i % N;
        long int m = (long int) (i-n)/N;

        sout[m+n*M] = sin[n+m*N];
    }
}

__global__ void
cfarSetWindow(hipFloatComplex *s, int windowLength, int guardInterval) {
    long int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= guardInterval && i < windowLength+guardInterval)
    {
        float winLen = (float)windowLength;
        s[i] = make_hipFloatComplex(1 / winLen, 0);
    }
}

__global__ void
rdSquareCopy(hipFloatComplex *sout, hipFloatComplex *sin, long int M, long int N) {
    long int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N*M)
    {
        float x = hipCabsf(sin[i]);
        sout[i] = make_hipFloatComplex(x*x,0);
    }
}

__global__ void
CFARComplexDivision(hipFloatComplex *rd, hipFloatComplex *cfar, long int M, long int N) {
    long int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N*M)
    {
        float cfsq = sqrt(2* hipCabsf(cfar[i]) / ((float)N));
        rd[i] = hipCdivf(rd[i], make_hipFloatComplex(cfsq,0));
    }
}

void matchedFilterProcessingCUDA_gpu(hipFloatComplex *signal, hipFloatComplex *waveform, hipFloatComplex *window, long int M, long int N) {
    size_t mem_size = sizeof(hipFloatComplex)*M*N;
    long int threadsPerBlock = 256;
    long int blocksPerGrid;

    // Allocate device memory for signal
    hipFloatComplex *d_signal, *d_waveform, *d_window;
    hipMalloc((void **)&d_signal, mem_size);
    hipMalloc((void **)&d_waveform, (mem_size/M));
    hipMalloc((void **)&d_window, (mem_size/M));

    // Copy host memory to device
    hipMemcpy(d_signal, signal, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(d_waveform, waveform, (mem_size/M), hipMemcpyHostToDevice);
    hipMemcpy(d_window, window, (mem_size/M), hipMemcpyHostToDevice);

    // Multiplying waveform with window
    blocksPerGrid =(N + threadsPerBlock - 1) / threadsPerBlock;
    rdComplexMultiply<<<blocksPerGrid,threadsPerBlock>>>(d_waveform, d_window, 1, N);

    // CUFFT plan simple API
    hipfftHandle plan;
    hipfftPlan1d(&plan, N, HIPFFT_C2C, M);

    // Performing device FFT
    hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_FORWARD);

    // Multiplying signal with waveform in fourier domain
    blocksPerGrid =(N*M + threadsPerBlock - 1) / threadsPerBlock;
    rdComplexMultiply<<<blocksPerGrid,threadsPerBlock>>>(d_signal, d_waveform, M, N);

    // Performing device IFFT
    hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_BACKWARD);

    // Copying data from device to host
    hipMemcpy(signal, d_signal, mem_size, hipMemcpyDeviceToHost);

    // Cleaning up
    hipfftDestroy(plan);
    hipFree(d_signal);
    hipFree(d_waveform);
    hipFree(d_window);
}

void dopplerProcessingCUDA_gpu(hipFloatComplex *signal, long int M, long N) {
    size_t mem_size = sizeof(hipFloatComplex)*M*N;

    // Allocate device memory for signal
    hipFloatComplex *d_signal;
    hipMalloc((void **)&d_signal, mem_size);
    // Copy host memory to device
    hipMemcpy(d_signal, signal, mem_size, hipMemcpyHostToDevice);

    // CUFFT plan simple API
    hipfftHandle plan;
    hipfftPlan1d(&plan, M, HIPFFT_C2C, N);

    // Performing device FFT
    hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_FORWARD);

    // Copying data from device to host
    hipMemcpy(signal, d_signal, mem_size, hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(d_signal);
}

void rangeDopplerProcessingCUDA_gpu(hipFloatComplex *signal, hipFloatComplex *waveform, hipFloatComplex *rangeWindow, hipFloatComplex *dopplerWindow, long int M, long int N) {
    size_t mem_size = sizeof(hipFloatComplex)*M*N;
    long int threadsPerBlock = 1024;//3584;
    long int blocksPerGrid;

    // Allocate device memory for signal
    hipFloatComplex *d_signal, *d_signal2, *d_waveform, *d_rangeWindow, *d_dopplerWindow;
    hipMalloc((void **)&d_signal, mem_size);
    hipMalloc((void **)&d_signal2, mem_size);
    hipMalloc((void **)&d_waveform, (mem_size/M));
    hipMalloc((void **)&d_rangeWindow, (mem_size/M));
    hipMalloc((void **)&d_dopplerWindow, (mem_size/N));

    // Copy host memory to device
    hipMemcpy(d_signal, signal, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(d_waveform, waveform, (mem_size/M), hipMemcpyHostToDevice);
    hipMemcpy(d_rangeWindow, rangeWindow, (mem_size/M), hipMemcpyHostToDevice);
    hipMemcpy(d_dopplerWindow, dopplerWindow, (mem_size/N), hipMemcpyHostToDevice);

    // Multiplying waveform with window
    blocksPerGrid =(N + threadsPerBlock - 1) / threadsPerBlock;
    rdComplexMultiply<<<blocksPerGrid,threadsPerBlock>>>(d_waveform, d_rangeWindow, 1, N);

    // CUFFT plan simple API
    hipfftHandle plan;
    hipfftPlan1d(&plan, N, HIPFFT_C2C, M);

    // Performing device FFT
    hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_FORWARD);

    // Multiplying signal with waveform in fourier domain
    blocksPerGrid =(N*M + threadsPerBlock - 1) / threadsPerBlock;
    rdComplexMultiply<<<blocksPerGrid,threadsPerBlock>>>(d_signal, d_waveform, M, N);

    // Performing device IFFT
    hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_BACKWARD);

    // Cleaning up plan
    hipfftDestroy(plan);

    // Transposing data
    blocksPerGrid =(N*M + threadsPerBlock - 1) / threadsPerBlock;
    rdComplexTranspose<<<blocksPerGrid,threadsPerBlock>>>(d_signal2, d_signal, M, N);

    // Multiplying signal with doppler window
    blocksPerGrid =(N*M + threadsPerBlock - 1) / threadsPerBlock;
    rdComplexMultiply<<<blocksPerGrid,threadsPerBlock>>>(d_signal2, d_dopplerWindow, N, M);

    // Doppler processing
    hipfftHandle plan2;
    hipfftPlan1d(&plan2, M, HIPFFT_C2C, N);

    // Performing device FFT
    hipfftExecC2C(plan2, (hipfftComplex *)d_signal2, (hipfftComplex *)d_signal2, HIPFFT_FORWARD);

    // Cleaning up plan
    hipfftDestroy(plan2);

    // Transposing data back
    blocksPerGrid =(N*M + threadsPerBlock - 1) / threadsPerBlock;
    rdComplexTranspose<<<blocksPerGrid,threadsPerBlock>>>(d_signal, d_signal2, N, M);

    // Copying data from device to host
    hipMemcpy(signal, d_signal, mem_size, hipMemcpyDeviceToHost);

    // Cleaning up
    hipFree(d_signal);
    hipFree(d_signal2);
    hipFree(d_waveform);
    hipFree(d_rangeWindow);
    hipFree(d_dopplerWindow);
}

void rangeDopplerCFARProcessingCUDA_gpu(hipFloatComplex *signal, hipFloatComplex *waveform, hipFloatComplex *rangeWindow, hipFloatComplex *dopplerWindow, long int M, long int N, int windowLength, int guardInterval) {
    size_t mem_size = sizeof(hipFloatComplex)*M*N;
    long int threadsPerBlock = 1024;//3584;
    long int blocksPerGrid;

    // Allocate device memory for signal
    hipFloatComplex *d_signal, *d_signal2, *d_waveform, *d_rangeWindow, *d_dopplerWindow;
    hipMalloc((void **)&d_signal, mem_size);
    hipMalloc((void **)&d_signal2, mem_size);
    hipMalloc((void **)&d_waveform, (mem_size/M));
    hipMalloc((void **)&d_rangeWindow, (mem_size/M));
    hipMalloc((void **)&d_dopplerWindow, (mem_size/N));

    // Copy host memory to device
    hipMemcpy(d_signal, signal, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(d_waveform, waveform, (mem_size/M), hipMemcpyHostToDevice);
    hipMemcpy(d_rangeWindow, rangeWindow, (mem_size/M), hipMemcpyHostToDevice);
    hipMemcpy(d_dopplerWindow, dopplerWindow, (mem_size/N), hipMemcpyHostToDevice);

    // Multiplying waveform with window
    blocksPerGrid =(N + threadsPerBlock - 1) / threadsPerBlock;
    rdComplexMultiply<<<blocksPerGrid,threadsPerBlock>>>(d_waveform, d_rangeWindow, 1, N);

    // CUFFT plan simple API
    hipfftHandle plan;
    hipfftPlan1d(&plan, N, HIPFFT_C2C, M);

    // Performing device FFT
    hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_FORWARD);

    // Multiplying signal with waveform in fourier domain
    blocksPerGrid =(N*M + threadsPerBlock - 1) / threadsPerBlock;
    rdComplexMultiply<<<blocksPerGrid,threadsPerBlock>>>(d_signal, d_waveform, M, N);

    // Performing device IFFT
    hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_BACKWARD);

    // Cleaning up plan
    hipfftDestroy(plan);

    // Transposing data
    blocksPerGrid =(N*M + threadsPerBlock - 1) / threadsPerBlock;
    rdComplexTranspose<<<blocksPerGrid,threadsPerBlock>>>(d_signal2, d_signal, M, N);

    // Multiplying signal with doppler window
    blocksPerGrid =(N*M + threadsPerBlock - 1) / threadsPerBlock;
    rdComplexMultiply<<<blocksPerGrid,threadsPerBlock>>>(d_signal2, d_dopplerWindow, N, M);

    // Doppler processing
    hipfftHandle plan2;
    hipfftPlan1d(&plan2, M, HIPFFT_C2C, N);

    // Performing device FFT
    hipfftExecC2C(plan2, (hipfftComplex *)d_signal2, (hipfftComplex *)d_signal2, HIPFFT_FORWARD);

    // Cleaning up plan
    hipfftDestroy(plan2);

    // Transposing data back
    blocksPerGrid =(N*M + threadsPerBlock - 1) / threadsPerBlock;
    rdComplexTranspose<<<blocksPerGrid,threadsPerBlock>>>(d_signal, d_signal2, N, M);

    // CFAR Processing
    // Making window
    hipMemset(d_waveform, 0, sizeof(hipFloatComplex)*N);
    blocksPerGrid =(windowLength+guardInterval + threadsPerBlock - 1) / threadsPerBlock;
    cfarSetWindow<<<blocksPerGrid,threadsPerBlock>>>(d_waveform, windowLength, guardInterval);

    // FFT of window
    hipfftHandle plan3;
    hipfftPlan1d(&plan3, N, HIPFFT_C2C, 1);

    hipfftExecC2C(plan3, (hipfftComplex *)d_waveform, (hipfftComplex *)d_waveform, HIPFFT_FORWARD);
    hipfftDestroy(plan3);

    // Copying and squaring RD matrix
    blocksPerGrid =(N*M + threadsPerBlock - 1) / threadsPerBlock;
    rdSquareCopy<<<blocksPerGrid,threadsPerBlock>>>(d_signal2, d_signal, M, N);

    // FFT of RD matrix
    hipfftHandle plan4;
    hipfftPlan1d(&plan4, N, HIPFFT_C2C, M);

    // Performing device FFT
    hipfftExecC2C(plan4, (hipfftComplex *)d_signal2, (hipfftComplex *)d_signal2, HIPFFT_FORWARD);

    // Multiplying RD Matrix in fourier domain with CFAR window
    blocksPerGrid =(N*M + threadsPerBlock - 1) / threadsPerBlock;
    rdComplexMultiply<<<blocksPerGrid,threadsPerBlock>>>(d_signal2, d_waveform, M, N);

    // Performing device IFFT
    hipfftExecC2C(plan4, (hipfftComplex *)d_signal2, (hipfftComplex *)d_signal2, HIPFFT_BACKWARD);
    hipfftDestroy(plan4);

    // Performing RD matrix elementwise division with CFAR matrix
    blocksPerGrid =(N*M + threadsPerBlock - 1) / threadsPerBlock;
    CFARComplexDivision<<<blocksPerGrid,threadsPerBlock>>>(d_signal, d_signal2, M, N);

    // Copying data from device to host
    hipMemcpy(signal, d_signal, mem_size, hipMemcpyDeviceToHost);

    // Cleaning up
    hipFree(d_signal);
    hipFree(d_signal2);
    hipFree(d_waveform);
    hipFree(d_rangeWindow);
    hipFree(d_dopplerWindow);
}
